#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>

//Device Code
__global__ void print_thread() {
    printf("x: %d  y: %d  z: %d \n", threadIdx.x, threadIdx.y, threadIdx.z);
}

//Host code
int main() {

    int nx, ny;

    nx = 16;
    ny = 16;

    //kernel launch parameters

    dim3 block(8, 8);
    dim3 grid(nx/block.x, ny/block.y);

    print_thread << <grid, block >> > (); // async call
    printf("Hello from CPU \n");
    hipDeviceSynchronize();// will make the prgram stall till all the launched kernels have finished execution


    hipDeviceReset();
    return 0;
}
