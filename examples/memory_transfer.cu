#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void mem_trs_test(int * input){
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    printf("tid: %d, gid : %d, value : %d \n", threadIdx.x,gid,input[gid]);
}

int main(){
    int size = 128;
    int byte_size = size * sizeof(int);
    int * h_input;

    h_input = (int*)malloc(byte_size);

    time_t t;
    srand((unsigned)time(&t));
    for(int i = 0; i < size; i++){
    h_input[i] = (int)(rand() & 0xff);
    }

    int *d_input;
    hipMalloc((void**)&d_input, byte_size);
    hipMemcpy(d_input, h_input,byte_size,hipMemcpyHostToDevice);

    dim3 block(64);
    dim3 grid(2);
    
    mem_trs_test <<<grid, block>>>(d_input);
    hipDeviceSynchronize();
    
    //reclaiming the memory
    hipFree(d_input);
    free(h_input);
    
    hipDeviceReset();
    return 0;
}