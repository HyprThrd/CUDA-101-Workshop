#include <stdio.h>

#include "hip/hip_runtime.h"


__global__ void hello_cuda(){
    printf("Hello from CUDA world \n");
}


int main(){
    
    dim3 block(8,2,1);
    dim3 threads_per_block(2,2,1);
    
    hello_cuda<<<block, threads_per_block>>>();

    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}
