#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

void query_device(){
  int deviceCount= 0;
  hipGetDeviceCount(&deviceCount);

  if(deviceCount == 0){
    printf("No CUDA Support Device found");
  }

  int devNo = 0;
  hipDeviceProp_t iProp;
  hipGetDeviceProperties( &iProp, devNo);

  printf("Device %d: %s\n", devNo, iProp.name);
  printf("Number of multiprocessors : %d\n ", iProp.multiProcessorCount);
  printf("Clock : %d \n", iProp.clockRate);
  printf("Compute Capability: %d.%d", iProp.major, iProp.minor);

}

int main(){
  query_device();
  return 0;
}