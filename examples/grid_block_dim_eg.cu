#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>

//Device Code
__global__ void print_details() {
    printf("blockIdx x: % d  y : % d  z : % d \nblockDim x: % d  y : % d  z : % d\ngridDim x: % d  y : % d  z : % d ", blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z,gridDim.x, gridDim.y, gridDim.z);
}

//Host code
int main() {

    int nx, ny;

    nx = 16;
    ny = 16;

    //kernel launch parameters

    dim3 block(8, 8);
    dim3 grid(nx/block.x, ny/block.y);

    print_details << <grid, block >> > (); // async call
    printf("Hello from CPU \n");
    hipDeviceSynchronize();// will make the prgram stall till all the launched kernels have finished execution


    hipDeviceReset();
    return 0;
}
