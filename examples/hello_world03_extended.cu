#include "hip/hip_runtime.h"

#include <stdio.h>

//Device Code
__global__ void hello_cuda(){
	printf("Hello from CUDA world %d \n", 10);
}

//Host code
int main(){ 
	//kernel launch parameters
  int nx, ny;
  nx = 16;
  ny = 4;
  
	dim3 block(8,2,1);
	dim3 grid(nx /block.x, ny/block.y);

	hello_cuda<<<grid, block>>>();
	hipDeviceSynchronize();// will make the prgram stall till all the launched kernels have finished execution
	hipDeviceReset();
	return 0;
}
