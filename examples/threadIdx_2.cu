#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void hello_cuda(){
    printf("Values: %d %d %d \n", threadIdx.x, threadIdx.y, threadIdx.z);
}

int main(){
    int nx,ny;
    nx = 8;
    ny = 1;

    dim3 block(2,1,1);
    dim3 threads(nx/block.x,ny/block.y);

    hello_cuda<<<block,threads>>>();
    hipDeviceSynchronize();
    hipDeviceReset();


    return 0;
}