#include "hip/hip_runtime.h"

#include <stdio.h>

//Device Code
__global__ void hello_cuda(){
	printf("Hello from CUDA world \n");
}

//Host code
int main(){ 
	//kernel launch parameters
	dim3 block(8,2,1);
	dim3 grid(2,2,1);

	hello_cuda<<<grid, block>>>();
	hipDeviceSynchronize();// will make the prgram stall till all the launched kernels have finished execution
	hipDeviceReset();
	return 0;
}
