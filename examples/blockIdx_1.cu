#include <stdio.h>

#include "hip/hip_runtime.h"



__global__ void print_block_id(){
    printf("%d %d %d \n", blockIdx.x, blockIdx.y, blockIdx.z);
}

int main(){
    int nx, ny;

    nx = 8;
    ny = 1;

    dim3 blocks(2,1,1);
    dim3 num_threads_block(nx/blocks.x, ny/blocks.y);

    print_block_id<<<blocks, num_threads_block>>>();

    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}