#include "hip/hip_runtime.h"

#include <stdio.h>


__global__ void hello_cuda(){
    printf("Hello from GPU\n");
}

int main(){
    hello_cuda<<<1, 10>>>();
    printf("I am here in CPU");

    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}
