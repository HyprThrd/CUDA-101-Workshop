#include "hip/hip_runtime.h"

#include <stdio.h>


__global__ void hello_cuda(){
    //Kernal code
}

int main(){

    //Kernal call with device launch params
    hello_cuda<<<1, 10>>>();
    //Device code

    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}
