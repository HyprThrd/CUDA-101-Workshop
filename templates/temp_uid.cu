#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>

//Device Code
__global__ void unique_idx_calc_threadIdx(int * input) {

int gid = ;

printf("threadIdx.x : %d, value : %d \n", gid, input[gid]);
}

//Host code
int main() {

    int array_size = 8;
    int array_byte_size = sizeof(int) * array_size;
    int h_data[] = {31, 34, 41, 44, 23, 32, 34, 23};

	for(int i = 0; i < array_size; i++){
		printf("%d ", h_data[i]);
	}
	printf("\n \n");

	int * d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 blocks(,);
	dim3 threads_per_block();
	unique_idx_calc_threadIdx <<<blocks, threads_per_block>>>(d_data);
	
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}
