#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

//Device Code
__global__ void hello_cuda(){
	//Kernal code
}

//Host code
int main(){ 
	//kernel launch parameters
  int nx, ny;
  nx = ;
  ny = ;
  
	dim3 block(,,);
	dim3 grid(nx /block.x, ny/block.y);

	hello_cuda<<<grid, block>>>();

	hipDeviceSynchronize();// will make the prgram stall till all the launched kernels have finished execution
	hipDeviceReset();
	return 0;
}
